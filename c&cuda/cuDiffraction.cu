#include "hip/hip_runtime.h"
// includes

#include "structures.h"
#include "cuprintf.cu"
#include "cudaComplex.cu"
#include "cudaVector.cu"
#include <string.h>
#include <math.h>
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef CUDAERR
#define CUDAERR
#ifdef __cplusplus
extern "C"
{
#endif
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "\n\nCuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
#ifdef __cplusplus
}
#endif
#endif
//#include <stdio.h>
// definitions
#define ATOM_TYPES 2
#define PI 3.14159265
// structures
#ifndef DATABLOCK
#define DATABLOCK
typedef struct 
{
	cPixel *pixels;
	int num_pix;
	cAtom *lattice;
	int num_atoms;
	int numZ;
	int *Z;
	float *allI;
} datablock;
#endif

// HOST FUNCTIONS
#ifdef __cplusplus
extern "C"
{
#endif
void initDev(int whichDevice) {
	hipSetDevice(whichDevice);
}
#ifdef __cplusplus
}
#endif
// DEVICE FUNCTIONS
#ifdef __cplusplus
extern "C"
{
#endif
void initDataBlock(datablock *data, cPixel *thePixels, int numPixels, cAtom *theLattice, int numAtoms, int *theZ, int num_Z, float *theAllI) {
	data->pixels = thePixels;
	data->num_pix = numPixels;
	data->lattice = theLattice;
	data->num_atoms = numAtoms;
	data->numZ = num_Z;
	data->Z = theZ;
	data->allI = theAllI;
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
__device__ float sumIntensities(float *allI, int numPixels) {
	float val = 0;
	for(int i = 0; i < numPixels; i++) {
		val += allI[i];
	}
	return val;
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
__global__ void diffraction_event(cPixel *cPixels, cComplex *sf, int num_pix, cAtom *lattice, int num_cAtoms, int *Z, int numZ)
{
	int pixIdx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(pixIdx < num_pix)
	{
		cuPrintf("\npixIdx: %d", pixIdx);
		cComplex temp_exp = {0, 0};
		cComplex temp_mult = {0, 0};
		cComplex temp_diffraction = {0, 0};
		for(int j = 0; j < numZ; j++) {
			// loop through the lattice and calc the scattering
			for(int i = 0; i < num_cAtoms; i++)
			{
				if(lattice[i].Z != Z[j]) { continue; }
				cuda_c_exp_imag(2*PI*cuda_v_dot(&lattice[i].v, &cPixels[pixIdx].q), &temp_exp);
				cuda_c_mult(&sf[pixIdx+j], &temp_exp, &temp_mult);
				cuda_c_add(&temp_diffraction, &temp_mult, &temp_diffraction);
			}
		}

		cPixels[pixIdx].I = temp_diffraction.re * temp_diffraction.re + temp_diffraction.im * temp_diffraction.im;
	}
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
__global__ void diffraction_event2(cPixel *cPixels, cComplex *sf, int num_pix, 
	cAtom *lattice, int num_atoms, int *Z, int numZ, float *I, int startingPixel)
{
	int pixIdx = threadIdx.x + blockIdx.x * blockDim.x + startingPixel;
	
	if(pixIdx < num_pix)
	{
		cPixels[pixIdx].temp2.re = 0;
		cPixels[pixIdx].temp2.im = 0;
		for(int j = 0; j < numZ; j++) {
			// loop through the lattice and calc the scattering
			//cuPrintf("\nsf_%d: %g, %g", j, sf[pixIdx+j].re, sf[pixIdx+j].im);
			for(int i = 0; i < num_atoms; i++)
			{
				if(lattice[i].Z != Z[j]) { continue; }
				cuda_c_exp_imag(2*PI*cuda_v_dot(&lattice[i].v, &cPixels[pixIdx].q), &cPixels[pixIdx].temp1);
				cuda_c_mult(&sf[pixIdx*numZ+j], &cPixels[pixIdx].temp1, &cPixels[pixIdx].temp1);
				cuda_c_add(&cPixels[pixIdx].temp2, &cPixels[pixIdx].temp1, &cPixels[pixIdx].temp2);
			}
		}
		I[pixIdx] = 10029;
		cPixels[pixIdx].I = cPixels[pixIdx].temp2.re * cPixels[pixIdx].temp2.re + cPixels[pixIdx].temp2.im * cPixels[pixIdx].temp2.im;
		/*cuPrintf("\npixIdx: %d, I: %g, v: %g, %g, %g q: %g, %g, %g", pixIdx, I[pixIdx], lattice[0].v.x, lattice[0].v.y, lattice[0].v.z,
			cPixels[pixIdx].q.x, cPixels[pixIdx].q.y, cPixels[pixIdx].q.z);*/
	}
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
__global__ void dev_event(int *a, int *b, int *c, int numToAdd) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx < numToAdd) {
		c[idx] = a[idx] + b[idx];
		//if(idx < 1) { cuPrintf("\ndev_event_1:%d", idx); }
	}
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
void printPixelsToFile(char *fName, cPixel *pixels, int numPixels, int numZ) {
	FILE *fp;
	int i, j;	
	fp = fopen(fName, "w");
	if(fp == NULL) {
		printf("\nFile: %s is not available for writing", fName);
		return;
	} else {
		printf("\nFile: %s opened", fName);
	}
	fprintf(fp, "\npixIdx: q.x, q.y, q.z\tI");
	for(i = 0; i < numPixels; i++) {
		fprintf(fp, "\n%d: %g, %g, %g\t%g", i, pixels[i].q.x, pixels[i].q.y,
								pixels[i].q.z, pixels[i].I);
		for(j = 0; j < numZ; j++) {
			fprintf(fp, "\n\t%d: %g, %g", j, pixels[i].sf[j].re, pixels[i].sf[j].im);
		}
	}
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
void cuEvent(cPixel *pixels, int numPixels, cAtom *lattice, int numAtoms, int *Z, int numZ, int iteration) {
	int *a, *b, *c, *dev_a, *dev_b, *dev_c, *dev_Z;
	int numToAdd, i;
	cPixel *dev_pixels;
	cComplex *dev_sf, *sf;
	cAtom *dev_lattice;
	float *dev_I, *I;
	int numThreadsPerLaunch;
	int numLoops;
	int numBlocks;
	int loopIdx;
	int threadsPerBlock;
	int cudaDeviceCount;
	int cudaDeviceToUse;
	int *numCudaCores;
	int maxNumCudaCores;
	hipDeviceProp_t deviceProp;
	
	/* INIT VARS */
	numToAdd = 100000;
	if(iteration == 0) { hipSetDevice(0); }

	/* ALLOCATE HOST MEMORY */
	a = (int *) malloc(sizeof(int) * numToAdd);
	b = (int *) malloc(sizeof(int) * numToAdd);
	c = (int *) malloc(sizeof(int) * numToAdd);
	sf = (cComplex *) malloc(sizeof(cComplex) * numZ * numPixels);
	I = (float *) malloc(sizeof(float) * numPixels);
	
	/* ALLOCATE DEVICE MEMORY */
	hipMalloc((void **) &dev_a, sizeof(int) * numToAdd);
	hipMalloc((void **) &dev_b, sizeof(int) * numToAdd);
	hipMalloc((void **) &dev_c, sizeof(int) * numToAdd);
	hipMalloc((void **) &dev_pixels, sizeof(cPixel) * numPixels);
	hipMalloc((void **) &dev_sf, sizeof(cComplex) * numZ * numPixels);
	hipMalloc((void **) &dev_lattice, sizeof(cAtom) * numAtoms);
	hipMalloc((void **) &dev_Z, sizeof(int) * numZ);
	hipMalloc((void **) &dev_I, sizeof(float) * numPixels);
	checkCUDAError("hipMalloc");
	
	//printPixelsToFile("Before copying to GPU", pixels, numPixels, numElemTypes);
	// fill the arrays
	for(i = 0; i < numToAdd; i++) {
		a[i] = i;
		b[i] = 2*i;
		c[i] = 0;
	}
	////printf("\nHello, World... from CUDA!");

	/* COPY DATA TO THE DEVICE */
	hipMemcpy(dev_a, a, sizeof(int) * numToAdd, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * numToAdd, hipMemcpyHostToDevice);
	hipMemcpy(dev_pixels, pixels, sizeof(cPixel) * numPixels, hipMemcpyHostToDevice);
	hipMemcpy(dev_lattice, lattice, sizeof(cAtom) * numAtoms, hipMemcpyHostToDevice);
	hipMemcpy(dev_Z, Z, sizeof(int) * numZ, hipMemcpyHostToDevice);
	////printf("\n\nBefore copying the scattering factors to a new array on the host: \n\n");
	for(int i = 0; i < numPixels; i++) {
		/*for(int j = 0; j < numZ; j++) {
			////printf("%g, %g\t", sf[i*numZ+j].re, sf[i*numZ+j].im);
			////printf("%g, %g\t", pixels[i].sf[j].re, pixels[i].sf[j].im);
		}*/
		for(int j = 0; j < numZ; j++) {
			memcpy(&sf[numZ*i+j], &pixels[i].sf[j], sizeof(cComplex));
		}
	}
	hipMemcpy(dev_sf, sf, sizeof(cComplex) * numZ * numPixels, hipMemcpyHostToDevice);
	
	/*//printf("\n\nAfter copying the scattering factors to a new array on the host: \n\n");
	for(int i = 0; i < 10; i++) {
		for(int j = 0; j < numZ; j++) {
			////printf("%g, %g\t", sf[i*numZ+j].re, sf[i*numZ+j].im);
			////printf("%g, %g\t", pixels[i].sf[j].re, pixels[i].sf[j].im);
		}
		////printf("\n");
	}
	hipMemcpy(sf, dev_sf, sizeof(cComplex) * numZ * numPixels, hipMemcpyDeviceToHost);
	////printf("\n\nAfter copying the scattering factors to the device and then back to the host: \n\n");
	for(int i = 0; i < 10; i++) {
		for(int j = 0; j < numZ; j++) {
			////printf("%g, %g\t", sf[i*numZ+j].re, sf[i*numZ+j].im);
			////printf("%g, %g\t", pixels[i].sf[j].re, pixels[i].sf[j].im);
		}
		////printf("\n");
	}
	////printf("\nnumZ: %d", numZ);*/
	
	
	
	checkCUDAError("memcpyToDevice");
	////printf("\nIn cuda, before kernel invocation, the first pixel's I = %g", pixels[0].I);
	// set up thread info
	//blocksPerGrid = numToAdd/threadsPerBlock+1;
	
	/*
	dev_event<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, numToAdd);	
	hipDeviceSynchronize();
	cudaPrintfDisplay(stdout, true);
	checkCUDAError("kernel invocation 1");
	cudaPrintfEnd();
	*/
	/////////////////////////////////////////////////////////////////////////
	// get cuda info to set the device as the one with the most cuda cores //
	/////////////////////////////////////////////////////////////////////////
	cudaDeviceCount = hipGetDeviceCount(&cudaDeviceCount);
	cudaDeviceToUse = 0;
	numCudaCores = (int *) malloc(sizeof(int) * cudaDeviceCount);
	maxNumCudaCores = 0;
	for(loopIdx = 0; loopIdx < cudaDeviceCount; loopIdx++) {
		hipGetDeviceProperties(&deviceProp, i);	// get the device properties with a specific cuda call
		numCudaCores[i] = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;	// get the number of cuda cores for device "i"
		if(numCudaCores[i] > maxNumCudaCores) {
			cudaDeviceToUse = i;
			maxNumCudaCores = numCudaCores[i];
		}
	}
	hipSetDevice(cudaDeviceToUse);
	
	///////////////////////////////////////////////////////////////////////////
	// launch a number of threads equal to the number of pixels to calculate //
	///////////////////////////////////////////////////////////////////////////
		
	numThreadsPerLaunch = numPixels;
	
	numLoops = (int) ceil((double) (numPixels / numThreadsPerLaunch));

	threadsPerBlock = 64;
		
	numBlocks = numThreadsPerLaunch / threadsPerBlock;
	
	cudaPrintfInit();
	
	for(loopIdx = 0; loopIdx < numLoops; loopIdx++) {
		diffraction_event2<<<numBlocks, threadsPerBlock>>>(dev_pixels, dev_sf, numPixels, dev_lattice, numAtoms, dev_Z, numZ, dev_I, numThreadsPerLaunch * loopIdx);
	}
		
	hipDeviceSynchronize();
	cudaPrintfDisplay(stdout, true);
	checkCUDAError("kernel invocation 2");
	
	hipMemcpy(pixels, dev_pixels, sizeof(cPixel) * numPixels, hipMemcpyDeviceToHost);
	hipMemcpy(I, dev_I, sizeof(float) * numPixels, hipMemcpyDeviceToHost);
	hipMemcpy(c, dev_c, sizeof(int) * numToAdd, hipMemcpyDeviceToHost);

	checkCUDAError("memcpyFromDevice");
	
	for(i = 0; i < numToAdd; i+=(numToAdd/3)) {
		////printf("\nresult: %d + %d = %d", a[i], b[i], c[i]);
	}
	free(a);
	free(b);
	free(c);
	free(sf);
	free(I);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_pixels);
	hipFree(dev_sf);
	hipFree(dev_lattice);
	hipFree(dev_Z);
	hipFree(dev_I);
	checkCUDAError("hipFree");
}
#ifdef __cplusplus
}
#endif
#ifdef __cplusplus
extern "C"
{
#endif
void displayDeviceProperties() {
	const int kb = 1024;
    const int mb = kb * kb;
    printf("\nNBody.GPU\n=========\n\n");

    printf("CUDA version:   v%d\n", CUDART_VERSION);    
    
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Devices: \n\n");

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%d: %s: %d.%d\n", i, props.name, props.major, props.minor);
        printf(" Global memory: %d mb\n", props.totalGlobalMem/mb);
		printf("Shared memory: %d kb\n", props.sharedMemPerBlock / kb);
		printf("Constant memory: %d kb\n", props.totalConstMem / kb);
        
		printf("  Block registers: %d\n", props.regsPerBlock);

        printf("  Warp size:          %d\n", props.warpSize);

		printf("  Threads per block: %d\n", props.maxThreadsPerBlock);
        
		printf("  Max block dimensions: [%d, %d, %d]\n", props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
        
		printf("  Max block dimensions: [%d, %d, %d]\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);
        
    }
}
#ifdef __cplusplus
}
#endif
