#include "hip/hip_runtime.h"
#if defined(CUCOMPLEX_CU)
#else
#define CUCOMPLEX_CU
/*** INCLUDES ***/
#include <stdlib.h>
#include <stdio.h>
/*** FUNCTIONS ***/

__device__ void cuda_c_add(cComplex *one, cComplex *two, cComplex *target)
{
	target->re = one->re + two->re;
	target->im = one->im + two->im;
}

__device__ void cuda_c_mult(cComplex *one, cComplex *two, cComplex *target)
{
	target->re = one->re * two->re - one->im * two->im;
	target->im = one->re * two->im + one->im * two->re;
}
__device__ void cuda_c_exp_cComplex(cComplex *c, cComplex *target)
{
	float ea = exp(c->re);
	target->re = ea * cos(c->im);
	target->im = ea * sin(c->im);
}

__device__ void cuda_c_exp_real(float d, cComplex *target)
{
	target->re = exp(d);
	target->im = 0;
}

__device__ void cuda_c_exp_imag(float im, cComplex *target)
{
	target->re = cos(im);
	target->im = sin(im);
}
__device__ void cuda_c_set(cComplex *target, float re, float im)
{
	target->re = re;
	target->im = im;
}
#endif