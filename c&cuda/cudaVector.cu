#include "hip/hip_runtime.h"
#if defined(CUVECTOR)
#else
#define CUVECTOR
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#define DEBUG 0

__device__ void cuda_v_add(cVector *v1, cVector *v2, cVector *target)
{
	target->x = (v1->x) + (v2->x);
	target->y = (v1->y) + (v2->y);
	target->z = (v1->z) + (v2->z);
}
__device__ void cuda_v_subtract(cVector *v1, cVector *v2, cVector *target)
{
	target->x = (v1->x) - (v2->x);
	target->y = (v1->y) - (v2->y);
	target->z = (v1->z) - (v2->z);
}

__device__ float cuda_v_dot(cVector *v1, cVector *v2)
{
	return (v1->x) * (v2->x) + (v1->y) * (v2->y) + (v1->z) * (v2->z);
}

__device__ void cuda_v_cross(cVector *v1, cVector *v2, cVector *target)
{
	target->x = (v1->y) * (v2->z) - (v1->z) * (v2->y);
	
	target->y = (v1->z) * (v2->x) - (v1->x) * (v2->z);
	
	target->z = (v1->x) * (v2->y) - (v1->y) * (v2->x);
}

__device__ void cuda_v_scale(cVector *v1, float scalar, cVector *target)
{
	(target->x) = (v1->x) * scalar;
	(target->y) = (v1->y) * scalar;
	(target->z) = (v1->z) * scalar;
}

__device__ float cuda_v_length(cVector *v)
{
	float length = sqrt((v->x)*(v->x) + (v->y)*(v->y) + (v->z)*(v->z));
	if(length == 0)
		return '\0';
	
	return length;
}

__device__ void cuda_v_abs(cVector *v, cVector *tmp)
{
	if(v->x < 0)
		tmp->x = -v->x;
	else
		tmp->x = v->x;

	if(v->y < 0)
		tmp->y = -v->y;
	else
		tmp->y = v->y;

	if(v->z < 0)
		tmp->z = -v->z;
	else
		tmp->z = v->z;
}
__device__ void cuda_v_unit(cVector *v, cVector *target)
{
	float temp = cuda_v_length(v);
	
	if(temp == 0)
	{
		(target->x) = 0;
		(target->y) = 0;
		(target->z) = 0;
	}
	else
	{
		(target->x) = (v->x)/temp;
		(target->y) = (v->y)/temp;
		(target->z) = (v->z)/temp;
	}
	
}

__device__ void cuda_v_set(cVector *target, float a, float b, float c)
{
	target->x = a;
	target->y = b;
	target->z = c;
}
#endif