#include "hip/hip_runtime.h"
// includes
#include "structures.h"
#include "cudaComplex.cu"
#include "cudaVector.cu"
#include "cVector.h"
#include "cComplex.h"
// definitions
#define PI 3.14159265
#define ATOM_TYPES 2
// structures
#ifndef CPIXEL
#define CPIXEL
typedef struct
{
	cVector q;
	cComplex *sf;
	float I;
} cPixel;
#endif

// HOST FUNCTIONS
void get_E_dependent_scattering(float wavelength, int Z, cComplex *target)
{
	target->re = (float)Z;
	target->im = 0.0;
}
void generate_f0(float q_len, int Z, cComplex *target, float elem_consts[10])
{
	int i;
	float f0 = 0;

	for(i = 1; i < 5; i++)
	{
		f0 += elem_consts[i] * exp(-1*elem_consts[i+4]*pow(q_len/(4*PI),2));
	}

	f0 += elem_consts[9];

	target->re = f0;
	target->im = 0;
}
void *init_cPixels(cVector *qx, cVector *qy, float qx_max, float qy_max, float q_step, int *Z, int num_cAtom_types, int *return_num_cPixels)
{
	int x_max, y_max, x_mid, y_mid, x, y, i, *ptr_Z, num_cPixels, count=0;
	cVector *x_temp, *y_temp;
	cComplex *e_dependent_sf, *ptr_e_dependent_sf, *f0_temp;
	cPixel *cPixels, *ptr_cPixels;
	float q_len;

	float c_consts[10] = {12.011, 2.31, 1.02, 1.589, 0.865, 20.844, 10.208, 0.569, 51.651, 0.216};
	float br_consts[10] = {79.904, 17.179, 5.236, 5.638, 3.985, 2.172, 16.58, 0.261, 41.433, 2.956};
	*return_num_cPixels = 0;
	
	x_temp = (cVector *)malloc(sizeof(cVector));
	if(x_temp == NULL)
		fprintf(stderr, "\nError allocating memory(%d)", __LINE__);

	y_temp = (cVector *)malloc(sizeof(cVector));
	if(y_temp == NULL)
		fprintf(stderr, "\nError allocating memory(%d)", __LINE__);

	e_dependent_sf = (cComplex *)malloc(num_cAtom_types * sizeof(cComplex));
	if(e_dependent_sf == NULL)
		fprintf(stderr, "\nError allocating memory(%d)", __LINE__);

	f0_temp = (cComplex *)malloc(sizeof(cComplex));
	if(f0_temp == NULL)
		fprintf(stderr, "\nError allocating memory(%d)", __LINE__);

	// calculate the number of cPixels in each direction
	// if the detector goes from -qxMax to qxMax then the number of cPixels in that direction
	// is (2*qxMax/qStep-1)
	x_mid = (int)ceil(qx_max/q_step);
	y_mid = (int)ceil(qy_max/q_step);

	x_max = 2*x_mid-1;
	y_max = 2*y_mid-1;
	
	num_cPixels = x_max * y_max;
	//printf("\ninit cPixels line: %d", __LINE__);
	ptr_Z = Z;
	ptr_e_dependent_sf = e_dependent_sf;
	// get the energy dependent structure factor for each scattering element
	for(i = 0; i < num_cAtom_types; i++, ptr_Z++, ptr_e_dependent_sf++)
	{
		get_E_dependent_scattering(.13702, *ptr_Z, ptr_e_dependent_sf);
	}
	//printf("\ninit cPixels line: %d", __LINE__);
	// allocate memory for the cPixels
	cPixels = (cPixel *)malloc(num_cPixels * sizeof(cPixel));
	if(cPixels == NULL)
		fprintf(stderr, "\nError allocating memory(%d)", __LINE__);
	ptr_cPixels = cPixels;
	//printf("\nSize of cPixel allocation: %d MB", (num_cPixels * (sizeof(cPixel) + num_cAtom_types * sizeof(cComplex)))/1000000);
	// loop through the cPixels
	for(x = 0; x < x_max; x++)
	{

		// calculate the q vector for this x value
		v_scale(qx, (float)(x-x_mid)*q_step, x_temp);		
		for(y = 0; y < x_max; y++)
		{
			// calculate the q vector for this y value
			v_scale(qy, (float)(y-y_mid)*q_step, y_temp);
			
			// sum the qx and qy vectors and store them in the cPixels->q vector
			v_add(x_temp, y_temp, &ptr_cPixels->q);

			ptr_Z = Z;
			ptr_e_dependent_sf = e_dependent_sf;
			q_len = v_length(&ptr_cPixels->q);
			for(i = 0; i < num_cAtom_types; i++, ptr_Z++, ptr_e_dependent_sf++)
			{
				////printf("\ninit cPixels line: %d", __LINE__);
				////printf("\n%d", *ptr_Z);
				// initialize the scattering factor to zero
				c_set(&ptr_cPixels->sf[i], 0, 0);
				// get the q-dependent portion of the scattering factor
				if(*ptr_Z == 6)
					generate_f0(q_len, *ptr_Z, f0_temp, c_consts);
				else if(*ptr_Z == 35)
					generate_f0(q_len, *ptr_Z, f0_temp, br_consts);
				// add the q-dependent portion of the scattering factor to the energy-dependent portion
				// and store it in the cPixel
				////printf("\ninit cPixels line: %d", __LINE__);
				c_add(f0_temp, ptr_e_dependent_sf, &ptr_cPixels->sf[i]);
			}
			ptr_cPixels->I = 0;
			count++;
			ptr_cPixels++;
		}
	}
	
	//printf("\nTotal cPixels initialized: %d", count);
	
	*return_num_cPixels = count;

	return cPixels;
}
void print_cPixels_file(cPixel *cPixels, float qx_max, float qy_max, float q_step, char *filename)
{
	int x_max, y_max, x_mid, y_mid, x, y, num_cPixels;
	FILE *fp, *pix;
	char *filename_pix = "pix.xray";
	cComplex *br, *c;

	cPixel *ptr_cPixels;
	// calculate the number of cPixels in each direction
	// if the detector goes from -qxMax to qxMax then the number of cPixels in that direction
	// is (2*qxMax/qStep-1)
	x_mid = (int)ceil(qx_max/q_step);
	y_mid = (int)ceil(qy_max/q_step);

	x_max = 2*x_mid-1;
	y_max = 2*y_mid-1;

	num_cPixels = x_max * y_max;
	//printf("\nnum cPixels: %d", num_cPixels);
	ptr_cPixels = cPixels;
	pix = fopen(filename_pix, "w");
	if(pix == NULL)
		//printf("\nProblem opening file(%d)", __LINE__);
	fp = fopen(filename, "w");
	if(fp == NULL)
		//printf("\nProblem opening file(%d)", __LINE__);
	fprintf(fp, "%d\n", 1);
	for(x = 0; x < x_max; x++)
	{
		for(y = 0; y < y_max; y++, ptr_cPixels++)
		{
			c = &ptr_cPixels->sf[0];
			br = &ptr_cPixels->sf[1];
			fprintf(fp, "%d\t%d\t%lf\n", x-x_mid, y-y_mid, ptr_cPixels->I);
			fprintf(pix, "%d\t%d\t%lf\n", x-x_mid, y-y_mid, ptr_cPixels->I);
			fprintf(pix, "%\t%lf\t%lf\t%lf\t\n", ptr_cPixels->q.x, ptr_cPixels->q.y, ptr_cPixels->q.z);
			fprintf(pix, "\tC:\t%lf\t%lf\n", c->re, c->im);
			fprintf(pix, "\tBr:\t%lf\t%lf\n", br->re, br->im);
			
		}
	}
	fflush(pix);
	fflush(fp);
	fclose(pix);
	fclose(fp);
}
void zero_cPixels(cPixel *current, int num_cPixels)
{
	int i;
	for(i = 0; i < num_cPixels; i++)
		current[i].I = 0;
}
void sum_diffraction(cPixel *current, cPixel *total, int num_cPixels)
{
	int i;
	for(i = 0; i < num_cPixels; i++)
		total[i].I += current[i].I;
}
void* convert_cTetrahedron_to_cAtoms(cTetrahedron *tetra_lattice, int num_tetra)
{
	int counter = 0;
	int i, j;
	cAtom *lattice;
	
	lattice = (cAtom *) malloc(sizeof(cAtom) * num_tetra*5);
	
	for(i = 0; i < num_tetra; i++)
		for(j = 0; j < 5; j++, counter++)
			lattice[counter] = tetra_lattice[i].cAtoms[j];
			
	return lattice;
}


// DEVICE FUNCTIONS
__global__ void scale_lattice(cAtom *dev_lattice, int num_cAtoms, float a)
{
	int cAtomIdx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(cAtomIdx < num_cAtoms)
		cuda_v_scale(&(dev_lattice[cAtomIdx].v), 1/a, &(dev_lattice[cAtomIdx].v));
}
__global__ void diffraction_event(cPixel *cPixels, cComplex *sf, int num_pix, cAtom *lattice, int num_cAtoms, char *Z, int numZ)
{
	int pixIdx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(pixIdx < num_pix)
	{
		cComplex temp_exp = {0, 0};
		cComplex temp_mult = {0, 0};
		cComplex temp_diffraction = {0, 0};
		
		for(int j = 0; j < numZ; j++) {
			// loop through the lattice and calc the scattering
			for(int i = 0; i < num_cAtoms; i++)
			{
				if(lattice[i].Z != Z[j]) { continue; }
				
				cuda_c_exp_imag(2*PI*cuda_v_dot(&lattice[i].v, &cPixels[pixIdx].q), &temp_exp);
				cuda_c_mult(&sf[pixIdx+j], &temp_exp, &temp_mult);
				cuda_c_add(&temp_diffraction, &temp_mult, &temp_diffraction);
			}
		}

		//temp_diffraction.re /= 1000000;
		//temp_diffraction.im /= 1000000;
		cPixels[pixIdx].I = temp_diffraction.re * temp_diffraction.re + temp_diffraction.im * temp_diffraction.im;
		//cur_pix->I = temp_diffraction.re;
		//cPixels[pixIdx].I = 5.;
	}
}
void diffraction(cPixel *pixels, cComplex *sf, int num_pix, cAtom *lattice, int num_cAtoms, char *Z, int numZ) {
	
}